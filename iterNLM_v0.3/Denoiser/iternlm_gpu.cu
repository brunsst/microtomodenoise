#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <chrono>
#include <fstream>

#include "iternlm_gpu.h"
#include "gpu_constants.cuh"
#include "../Geometry/hdcommunication.h"
#include "../Geometry/auxiliary.h"
#include "iternlm_prepare.h"

/*********************************************************************************************************************************************************
 * Location: Helmholtz-Zentrum fuer Material und Kuestenforschung, Max-Planck-Strasse 1, 21502 Geesthacht
 * Author: Stefan Bruns
 * Contact: bruns@nano.ku.dk
 *
 * License: TBA
 *********************************************************************************************************************************************************/

namespace denoise
{
	namespace gpu_denoise
	{
		__device__ __inline__ float expapproximation(float x){return (120.f + 60.f*x + 12.f*x*x + x*x*x)/(120.f - 60.f*x + 12.f*x*x - x*x*x);} //3rd order Pade approximation

		__global__ void pad_reflective(float *input, float *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx0 = gpu_const::nx;
			int ny0 = gpu_const::ny;
			int nz0 = gpu_const::nz;

			idx_type nslice0 = nx0*ny0;

			int padding0 = gpu_const::padding[0];
			int padding1 = gpu_const::padding[1];
			int padding2 = gpu_const::padding[2];

			int nx1 = nx0 + padding0 + gpu_const::padding[3];
			int ny1 = ny0 + padding1 + gpu_const::padding[4];
			int nz1 = nz0 + padding2 + gpu_const::padding[5];

			idx_type nslice1 = nx1*ny1;
			idx_type nstack1 = nz1*nslice1;

			idx_type idx1 = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx1 >= nstack1) idx1 = threadIdx.x;

			int z1 = idx1/nslice1;
			int y1 = (idx1-z1*nslice1)/nx1;
			int x1 = idx1-z1*nslice1-y1*nx1;

			int z0 = z1-padding2;
			int y0 = y1-padding1;
			int x0 = x1-padding0;

			while (z0 < 0 || z0 >= nz0 || y0 < 0 || y0 >= ny0 || x0 < 0 || x0 >= nx0)
			{
				if (z0 < 0) z0 = -z0;
				if (y0 < 0) y0 = -y0;
				if (x0 < 0) x0 = -x0;

				if (z0 >= nz0) z0 = 2*nz0-z0-2;
				if (y0 >= ny0) y0 = 2*ny0-y0-2;
				if (x0 >= nx0) x0 = 2*nx0-x0-2;
			}

			long long int idx0 = z0*nslice0+y0*nx0+x0;

			__syncthreads();
			/////////////////////////////////////////////

			output[idx1] = input[idx0];

			return;
		}

		__global__ void apply_filter_generic(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions, float *distweight)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			int nsize_patch = gpu_const::nsize_patch;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];
			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;
				float tmp = image_previous[idx1]-image_previous[idx0];
				distance += (tmp*tmp)*distweight[0];

				#pragma unroll
				for (int p = 1; p < nsize_patch; p++)
				{
					tmp = image_previous[idx1 + patch_positions[p]]-image_previous[idx0 + patch_positions[p]];
					distance += (tmp*tmp)*distweight[p];
				}
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch111(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1 =  patch_positions[1];
			idx_type ppos2 =  patch_positions[2];
			idx_type ppos3 =  patch_positions[3];
			idx_type ppos4 =  patch_positions[4];
			idx_type ppos5 =  patch_positions[5];
			idx_type ppos6 =  patch_positions[6];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0];
			float val_orig1 = image_previous[idx0 + ppos1];
			float val_orig2 = image_previous[idx0 + ppos2];
			float val_orig3 = image_previous[idx0 + ppos3];
			float val_orig4 = image_previous[idx0 + ppos4];
			float val_orig5 = image_previous[idx0 + ppos5];
			float val_orig6 = image_previous[idx0 + ppos6];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.142857143f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.142857143f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;

				//float this_weight = (distance > -3.56648f) ? expapproximation(distance) : 0.0f;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch112(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1= patch_positions[1];idx_type ppos2= patch_positions[2];idx_type ppos3= patch_positions[3];idx_type ppos4= patch_positions[4];
			idx_type ppos5= patch_positions[5];idx_type ppos6= patch_positions[6];idx_type ppos7= patch_positions[7];idx_type ppos8= patch_positions[8];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ];
			float val_orig1=image_previous[idx0+ppos1];float val_orig2=image_previous[idx0+ppos2];
			float val_orig3=image_previous[idx0+ppos3];float val_orig4=image_previous[idx0+ppos4];
			float val_orig5=image_previous[idx0+ppos5];float val_orig6=image_previous[idx0+ppos6];float val_orig7=image_previous[idx0+ppos7];
			float val_orig8=image_previous[idx0+ppos8];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.04f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch113(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1= patch_positions[1];idx_type ppos2= patch_positions[2];idx_type ppos3= patch_positions[3];idx_type ppos4= patch_positions[4];
			idx_type ppos5= patch_positions[5];idx_type ppos6= patch_positions[6];idx_type ppos7= patch_positions[7];idx_type ppos8= patch_positions[8];
			idx_type ppos9= patch_positions[9];idx_type ppos10= patch_positions[10];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ];float val_orig1=image_previous[idx0+ppos1];
			float val_orig2=image_previous[idx0+ppos2];float val_orig3=image_previous[idx0+ppos3];float val_orig4=image_previous[idx0+ppos4];
			float val_orig5=image_previous[idx0+ppos5];float val_orig6=image_previous[idx0+ppos6];
			float val_orig7=image_previous[idx0+ppos7];float val_orig8=image_previous[idx0+ppos8];
			float val_orig9=image_previous[idx0+ppos9];float val_orig10=image_previous[idx0+ppos10];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.0204082f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch221(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1 =  patch_positions[1];
			idx_type ppos2 =  patch_positions[2];
			idx_type ppos3 =  patch_positions[3];
			idx_type ppos4 =  patch_positions[4];
			idx_type ppos5 =  patch_positions[5];
			idx_type ppos6 =  patch_positions[6];
			idx_type ppos7 =  patch_positions[7];
			idx_type ppos8 =  patch_positions[8];
			idx_type ppos9 =  patch_positions[9];
			idx_type ppos10=  patch_positions[10];
			idx_type ppos11=  patch_positions[11];
			idx_type ppos12=  patch_positions[12];
			idx_type ppos13=  patch_positions[13];
			idx_type ppos14=  patch_positions[14];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ];
			float val_orig1 = image_previous[idx0 + ppos1];
			float val_orig2 = image_previous[idx0 + ppos2];
			float val_orig3 = image_previous[idx0 + ppos3];
			float val_orig4 = image_previous[idx0 + ppos4];
			float val_orig5 = image_previous[idx0 + ppos5];
			float val_orig6 = image_previous[idx0 + ppos6];
			float val_orig7 = image_previous[idx0 + ppos7];
			float val_orig8 = image_previous[idx0 + ppos8];
			float val_orig9 = image_previous[idx0 + ppos9];
			float val_orig10= image_previous[idx0 + ppos10];
			float val_orig11= image_previous[idx0 + ppos11];
			float val_orig12= image_previous[idx0 + ppos12];
			float val_orig13= image_previous[idx0 + ppos13];
			float val_orig14= image_previous[idx0 + ppos14];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos11]-val_orig11; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos12]-val_orig12; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos13]-val_orig13; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos14]-val_orig14; distance += (tmp*tmp)*0.111111f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch222(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1 =  patch_positions[1]; idx_type ppos2 =  patch_positions[2];
			idx_type ppos3 =  patch_positions[3]; idx_type ppos4 =  patch_positions[4];
			idx_type ppos5 =  patch_positions[5]; idx_type ppos6 =  patch_positions[6];
			idx_type ppos7 =  patch_positions[7]; idx_type ppos8 =  patch_positions[8];
			idx_type ppos9 =  patch_positions[9]; idx_type ppos10=  patch_positions[10];
			idx_type ppos11=  patch_positions[11];idx_type ppos12=  patch_positions[12];
			idx_type ppos13=  patch_positions[13];idx_type ppos14=  patch_positions[14];
			idx_type ppos15=  patch_positions[15];idx_type ppos16=  patch_positions[16];
			idx_type ppos17=  patch_positions[17];idx_type ppos18=  patch_positions[18];
			idx_type ppos19=  patch_positions[19];idx_type ppos20=  patch_positions[20];
			idx_type ppos21=  patch_positions[21];idx_type ppos22=  patch_positions[22];
			idx_type ppos23=  patch_positions[23];idx_type ppos24=  patch_positions[24];
			idx_type ppos25=  patch_positions[25];idx_type ppos26=  patch_positions[26];
			idx_type ppos27=  patch_positions[27];idx_type ppos28=  patch_positions[28];
			idx_type ppos29=  patch_positions[29];idx_type ppos30=  patch_positions[30];
			idx_type ppos31=  patch_positions[31];idx_type ppos32=  patch_positions[32];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ]; float val_orig1 = image_previous[idx0 + ppos1];
			float val_orig2 = image_previous[idx0 + ppos2]; float val_orig3 = image_previous[idx0 + ppos3];
			float val_orig4 = image_previous[idx0 + ppos4]; float val_orig5 = image_previous[idx0 + ppos5];
			float val_orig6 = image_previous[idx0 + ppos6]; float val_orig7 = image_previous[idx0 + ppos7];
			float val_orig8 = image_previous[idx0 + ppos8]; float val_orig9 = image_previous[idx0 + ppos9];
			float val_orig10= image_previous[idx0 + ppos10];float val_orig11= image_previous[idx0 + ppos11];
			float val_orig12= image_previous[idx0 + ppos12];float val_orig13= image_previous[idx0 + ppos13];
			float val_orig14= image_previous[idx0 + ppos14];float val_orig15= image_previous[idx0 + ppos15];
			float val_orig16= image_previous[idx0 + ppos16];float val_orig17= image_previous[idx0 + ppos17];
			float val_orig18= image_previous[idx0 + ppos18];float val_orig19= image_previous[idx0 + ppos19];
			float val_orig20= image_previous[idx0 + ppos20];float val_orig21= image_previous[idx0 + ppos21];
			float val_orig22= image_previous[idx0 + ppos22];float val_orig23= image_previous[idx0 + ppos23];
			float val_orig24= image_previous[idx0 + ppos24];float val_orig25= image_previous[idx0 + ppos25];
			float val_orig26= image_previous[idx0 + ppos26];float val_orig27= image_previous[idx0 + ppos27];
			float val_orig28= image_previous[idx0 + ppos28];float val_orig29= image_previous[idx0 + ppos29];
			float val_orig30= image_previous[idx0 + ppos30];float val_orig31= image_previous[idx0 + ppos31];
			float val_orig32= image_previous[idx0 + ppos32];


			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos11]-val_orig11; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos12]-val_orig12; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos13]-val_orig13; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos14]-val_orig14; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos15]-val_orig15; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos16]-val_orig16; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos17]-val_orig17; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos18]-val_orig18; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos19]-val_orig19; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos20]-val_orig20; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos21]-val_orig21; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos22]-val_orig22; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos23]-val_orig23; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos24]-val_orig24; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos25]-val_orig25; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos26]-val_orig26; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos27]-val_orig27; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos28]-val_orig28; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos29]-val_orig29; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos30]-val_orig30; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos31]-val_orig31; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos32]-val_orig32; distance += (tmp*tmp)*0.04f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch331(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1 =  patch_positions[1]; idx_type ppos2 =  patch_positions[2];
			idx_type ppos3 =  patch_positions[3]; idx_type ppos4 =  patch_positions[4];
			idx_type ppos5 =  patch_positions[5]; idx_type ppos6 =  patch_positions[6];
			idx_type ppos7 =  patch_positions[7]; idx_type ppos8 =  patch_positions[8];
			idx_type ppos9 =  patch_positions[9]; idx_type ppos10=  patch_positions[10];
			idx_type ppos11=  patch_positions[11];idx_type ppos12=  patch_positions[12];
			idx_type ppos13=  patch_positions[13];idx_type ppos14=  patch_positions[14];
			idx_type ppos15=  patch_positions[15];idx_type ppos16=  patch_positions[16];
			idx_type ppos17=  patch_positions[17];idx_type ppos18=  patch_positions[18];
			idx_type ppos19=  patch_positions[19];idx_type ppos20=  patch_positions[20];
			idx_type ppos21=  patch_positions[21];idx_type ppos22=  patch_positions[22];
			idx_type ppos23=  patch_positions[23];idx_type ppos24=  patch_positions[24];
			idx_type ppos25=  patch_positions[25];idx_type ppos26=  patch_positions[26];
			idx_type ppos27=  patch_positions[27];idx_type ppos28=  patch_positions[28];
			idx_type ppos29=  patch_positions[29];idx_type ppos30=  patch_positions[30];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ]; float val_orig1 = image_previous[idx0 + ppos1];
			float val_orig2 = image_previous[idx0 + ppos2]; float val_orig3 = image_previous[idx0 + ppos3];
			float val_orig4 = image_previous[idx0 + ppos4]; float val_orig5 = image_previous[idx0 + ppos5];
			float val_orig6 = image_previous[idx0 + ppos6]; float val_orig7 = image_previous[idx0 + ppos7];
			float val_orig8 = image_previous[idx0 + ppos8]; float val_orig9 = image_previous[idx0 + ppos9];
			float val_orig10= image_previous[idx0 + ppos10];float val_orig11= image_previous[idx0 + ppos11];
			float val_orig12= image_previous[idx0 + ppos12];float val_orig13= image_previous[idx0 + ppos13];
			float val_orig14= image_previous[idx0 + ppos14];float val_orig15= image_previous[idx0 + ppos15];
			float val_orig16= image_previous[idx0 + ppos16];float val_orig17= image_previous[idx0 + ppos17];
			float val_orig18= image_previous[idx0 + ppos18];float val_orig19= image_previous[idx0 + ppos19];
			float val_orig20= image_previous[idx0 + ppos20];float val_orig21= image_previous[idx0 + ppos21];
			float val_orig22= image_previous[idx0 + ppos22];float val_orig23= image_previous[idx0 + ppos23];
			float val_orig24= image_previous[idx0 + ppos24];float val_orig25= image_previous[idx0 + ppos25];
			float val_orig26= image_previous[idx0 + ppos26];float val_orig27= image_previous[idx0 + ppos27];
			float val_orig28= image_previous[idx0 + ppos28];float val_orig29= image_previous[idx0 + ppos29];
			float val_orig30= image_previous[idx0 + ppos30];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos11]-val_orig11; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos12]-val_orig12; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos13]-val_orig13; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos14]-val_orig14; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos15]-val_orig15; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos16]-val_orig16; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos17]-val_orig17; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos18]-val_orig18; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos19]-val_orig19; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos20]-val_orig20; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos21]-val_orig21; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos22]-val_orig22; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos23]-val_orig23; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos24]-val_orig24; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos25]-val_orig25; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos26]-val_orig26; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos27]-val_orig27; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos28]-val_orig28; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos29]-val_orig29; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos30]-val_orig30; distance += (tmp*tmp)*0.111111f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch332(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			idx_type ppos1= patch_positions[1];idx_type ppos2= patch_positions[2];idx_type ppos3= patch_positions[3];idx_type ppos4= patch_positions[4];
			idx_type ppos5= patch_positions[5];idx_type ppos6= patch_positions[6];idx_type ppos7= patch_positions[7];idx_type ppos8= patch_positions[8];
			idx_type ppos9= patch_positions[9];idx_type ppos10= patch_positions[10];idx_type ppos11= patch_positions[11];idx_type ppos12= patch_positions[12];
			idx_type ppos13= patch_positions[13];idx_type ppos14= patch_positions[14];idx_type ppos15= patch_positions[15];idx_type ppos16= patch_positions[16];
			idx_type ppos17= patch_positions[17];idx_type ppos18= patch_positions[18];idx_type ppos19= patch_positions[19];idx_type ppos20= patch_positions[20];
			idx_type ppos21= patch_positions[21];idx_type ppos22= patch_positions[22];idx_type ppos23= patch_positions[23];idx_type ppos24= patch_positions[24];
			idx_type ppos25= patch_positions[25];idx_type ppos26= patch_positions[26];idx_type ppos27= patch_positions[27];idx_type ppos28= patch_positions[28];
			idx_type ppos29= patch_positions[29];idx_type ppos30= patch_positions[30];idx_type ppos31= patch_positions[31];idx_type ppos32= patch_positions[32];
			idx_type ppos33= patch_positions[33];idx_type ppos34= patch_positions[34];idx_type ppos35= patch_positions[35];idx_type ppos36= patch_positions[36];
			idx_type ppos37= patch_positions[37];idx_type ppos38= patch_positions[38];idx_type ppos39= patch_positions[39];idx_type ppos40= patch_positions[40];
			idx_type ppos41= patch_positions[41];idx_type ppos42= patch_positions[42];idx_type ppos43= patch_positions[43];idx_type ppos44= patch_positions[44];
			idx_type ppos45= patch_positions[45];idx_type ppos46= patch_positions[46];idx_type ppos47= patch_positions[47];idx_type ppos48= patch_positions[48];
			idx_type ppos49= patch_positions[49];idx_type ppos50= patch_positions[50];idx_type ppos51= patch_positions[51];idx_type ppos52= patch_positions[52];
			idx_type ppos53= patch_positions[53];idx_type ppos54= patch_positions[54];idx_type ppos55= patch_positions[55];idx_type ppos56= patch_positions[56];
			idx_type ppos57= patch_positions[57];idx_type ppos58= patch_positions[58];idx_type ppos59= patch_positions[59];idx_type ppos60= patch_positions[60];
			idx_type ppos61= patch_positions[61];idx_type ppos62= patch_positions[62];idx_type ppos63= patch_positions[63];idx_type ppos64= patch_positions[64];
			idx_type ppos65= patch_positions[65];idx_type ppos66= patch_positions[66];idx_type ppos67= patch_positions[67];idx_type ppos68= patch_positions[68];
			idx_type ppos69= patch_positions[69];idx_type ppos70= patch_positions[70];idx_type ppos71= patch_positions[71];idx_type ppos72= patch_positions[72];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ];float val_orig1=image_previous[idx0+ppos1];float val_orig2=image_previous[idx0+ppos2];float val_orig3=image_previous[idx0+ppos3];float val_orig4=image_previous[idx0+ppos4];
			float val_orig5=image_previous[idx0+ppos5];float val_orig6=image_previous[idx0+ppos6];float val_orig7=image_previous[idx0+ppos7];float val_orig8=image_previous[idx0+ppos8];
			float val_orig9=image_previous[idx0+ppos9];float val_orig10=image_previous[idx0+ppos10];float val_orig11=image_previous[idx0+ppos11];float val_orig12=image_previous[idx0+ppos12];
			float val_orig13=image_previous[idx0+ppos13];float val_orig14=image_previous[idx0+ppos14];float val_orig15=image_previous[idx0+ppos15];float val_orig16=image_previous[idx0+ppos16];
			float val_orig17=image_previous[idx0+ppos17];float val_orig18=image_previous[idx0+ppos18];float val_orig19=image_previous[idx0+ppos19];float val_orig20=image_previous[idx0+ppos20];
			float val_orig21=image_previous[idx0+ppos21];float val_orig22=image_previous[idx0+ppos22];float val_orig23=image_previous[idx0+ppos23];float val_orig24=image_previous[idx0+ppos24];
			float val_orig25=image_previous[idx0+ppos25];float val_orig26=image_previous[idx0+ppos26];float val_orig27=image_previous[idx0+ppos27];float val_orig28=image_previous[idx0+ppos28];
			float val_orig29=image_previous[idx0+ppos29];float val_orig30=image_previous[idx0+ppos30];float val_orig31=image_previous[idx0+ppos31];float val_orig32=image_previous[idx0+ppos32];
			float val_orig33=image_previous[idx0+ppos33];float val_orig34=image_previous[idx0+ppos34];float val_orig35=image_previous[idx0+ppos35];float val_orig36=image_previous[idx0+ppos36];
			float val_orig37=image_previous[idx0+ppos37];float val_orig38=image_previous[idx0+ppos38];float val_orig39=image_previous[idx0+ppos39];float val_orig40=image_previous[idx0+ppos40];
			float val_orig41=image_previous[idx0+ppos41];float val_orig42=image_previous[idx0+ppos42];float val_orig43=image_previous[idx0+ppos43];float val_orig44=image_previous[idx0+ppos44];
			float val_orig45=image_previous[idx0+ppos45];float val_orig46=image_previous[idx0+ppos46];float val_orig47=image_previous[idx0+ppos47];float val_orig48=image_previous[idx0+ppos48];
			float val_orig49=image_previous[idx0+ppos49];float val_orig50=image_previous[idx0+ppos50];float val_orig51=image_previous[idx0+ppos51];float val_orig52=image_previous[idx0+ppos52];
			float val_orig53=image_previous[idx0+ppos53];float val_orig54=image_previous[idx0+ppos54];float val_orig55=image_previous[idx0+ppos55];float val_orig56=image_previous[idx0+ppos56];
			float val_orig57=image_previous[idx0+ppos57];float val_orig58=image_previous[idx0+ppos58];float val_orig59=image_previous[idx0+ppos59];float val_orig60=image_previous[idx0+ppos60];
			float val_orig61=image_previous[idx0+ppos61];float val_orig62=image_previous[idx0+ppos62];float val_orig63=image_previous[idx0+ppos63];float val_orig64=image_previous[idx0+ppos64];
			float val_orig65=image_previous[idx0+ppos65];float val_orig66=image_previous[idx0+ppos66];float val_orig67=image_previous[idx0+ppos67];float val_orig68=image_previous[idx0+ppos68];
			float val_orig69=image_previous[idx0+ppos69];float val_orig70=image_previous[idx0+ppos70];float val_orig71=image_previous[idx0+ppos71];float val_orig72=image_previous[idx0+ppos72];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos11]-val_orig11; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos12]-val_orig12; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos13]-val_orig13; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos14]-val_orig14; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos15]-val_orig15; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos16]-val_orig16; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos17]-val_orig17; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos18]-val_orig18; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos19]-val_orig19; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos20]-val_orig20; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos21]-val_orig21; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos22]-val_orig22; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos23]-val_orig23; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos24]-val_orig24; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos25]-val_orig25; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos26]-val_orig26; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos27]-val_orig27; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos28]-val_orig28; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos29]-val_orig29; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos30]-val_orig30; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos31]-val_orig31; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos32]-val_orig32; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos33]-val_orig33; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos34]-val_orig34; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos35]-val_orig35; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos36]-val_orig36; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos37]-val_orig37; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos38]-val_orig38; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos39]-val_orig39; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos40]-val_orig40; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos41]-val_orig41; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos42]-val_orig42; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos43]-val_orig43; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos44]-val_orig44; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos45]-val_orig45; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos46]-val_orig46; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos47]-val_orig47; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos48]-val_orig48; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos49]-val_orig49; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos50]-val_orig50; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos51]-val_orig51; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos52]-val_orig52; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos53]-val_orig53; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos54]-val_orig54; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos55]-val_orig55; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos56]-val_orig56; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos57]-val_orig57; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos58]-val_orig58; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos59]-val_orig59; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos60]-val_orig60; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos61]-val_orig61; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos62]-val_orig62; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos63]-val_orig63; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos64]-val_orig64; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos65]-val_orig65; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos66]-val_orig66; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos67]-val_orig67; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos68]-val_orig68; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos69]-val_orig69; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos70]-val_orig70; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos71]-val_orig71; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos72]-val_orig72; distance += (tmp*tmp)*0.04f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
		__global__ void apply_filter_patch333(float *image_raw, float *image_previous, float *next_result, float *sigma_list, idx_type *search_positions, idx_type *patch_positions)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx;
			int ny = gpu_const::ny;
			int nz = gpu_const::nz;

			int xpad = gpu_const::padding[0];
			int ypad = gpu_const::padding[1];
			int zpad = gpu_const::padding[2];

			int nsize_search = gpu_const::nsize_search;
			float beta = gpu_const::beta;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			int nx_padded = nx+2*xpad;
			idx_type nslice_padded = nx_padded*(ny+2*ypad);

			idx_type idx_unpadded = (blockIdx.x*blockDim.x+threadIdx.x); //idx without padding
			if (idx_unpadded >= nstack) {idx_unpadded = threadIdx.x;}

			int z0 = idx_unpadded/nslice;
			int y0 = (idx_unpadded-z0*nslice)/nx;
			int x0 = idx_unpadded-z0*nslice-y0*nx;

			z0 += zpad;
			y0 += ypad;
			x0 += xpad;

			idx_type idx0 = z0*nslice_padded + y0*nx_padded + x0; //idx_padded

			float filtervalue = 0.0f;
			float filterweight = 0.0f;
			float maxweight = 0.0f;

			//float values_origin[115]; static memory is the second best choice
			/////////////////////////////////////////////

			__syncthreads();
			float noisy_value_origin = image_raw[idx0];

			//Putting this into constant memory doesn't help much
			idx_type ppos1= patch_positions[1];idx_type ppos2= patch_positions[2];idx_type ppos3= patch_positions[3];idx_type ppos4= patch_positions[4];
			idx_type ppos5= patch_positions[5];idx_type ppos6= patch_positions[6];idx_type ppos7= patch_positions[7];idx_type ppos8= patch_positions[8];
			idx_type ppos9= patch_positions[9];idx_type ppos10= patch_positions[10];idx_type ppos11= patch_positions[11];idx_type ppos12= patch_positions[12];
			idx_type ppos13= patch_positions[13];idx_type ppos14= patch_positions[14];idx_type ppos15= patch_positions[15];idx_type ppos16= patch_positions[16];
			idx_type ppos17= patch_positions[17];idx_type ppos18= patch_positions[18];idx_type ppos19= patch_positions[19];idx_type ppos20= patch_positions[20];
			idx_type ppos21= patch_positions[21];idx_type ppos22= patch_positions[22];idx_type ppos23= patch_positions[23];idx_type ppos24= patch_positions[24];
			idx_type ppos25= patch_positions[25];idx_type ppos26= patch_positions[26];idx_type ppos27= patch_positions[27];idx_type ppos28= patch_positions[28];
			idx_type ppos29= patch_positions[29];idx_type ppos30= patch_positions[30];idx_type ppos31= patch_positions[31];idx_type ppos32= patch_positions[32];
			idx_type ppos33= patch_positions[33];idx_type ppos34= patch_positions[34];idx_type ppos35= patch_positions[35];idx_type ppos36= patch_positions[36];
			idx_type ppos37= patch_positions[37];idx_type ppos38= patch_positions[38];idx_type ppos39= patch_positions[39];idx_type ppos40= patch_positions[40];
			idx_type ppos41= patch_positions[41];idx_type ppos42= patch_positions[42];idx_type ppos43= patch_positions[43];idx_type ppos44= patch_positions[44];
			idx_type ppos45= patch_positions[45];idx_type ppos46= patch_positions[46];idx_type ppos47= patch_positions[47];idx_type ppos48= patch_positions[48];
			idx_type ppos49= patch_positions[49];idx_type ppos50= patch_positions[50];idx_type ppos51= patch_positions[51];idx_type ppos52= patch_positions[52];
			idx_type ppos53= patch_positions[53];idx_type ppos54= patch_positions[54];idx_type ppos55= patch_positions[55];idx_type ppos56= patch_positions[56];
			idx_type ppos57= patch_positions[57];idx_type ppos58= patch_positions[58];idx_type ppos59= patch_positions[59];idx_type ppos60= patch_positions[60];
			idx_type ppos61= patch_positions[61];idx_type ppos62= patch_positions[62];idx_type ppos63= patch_positions[63];idx_type ppos64= patch_positions[64];
			idx_type ppos65= patch_positions[65];idx_type ppos66= patch_positions[66];idx_type ppos67= patch_positions[67];idx_type ppos68= patch_positions[68];
			idx_type ppos69= patch_positions[69];idx_type ppos70= patch_positions[70];idx_type ppos71= patch_positions[71];idx_type ppos72= patch_positions[72];
			idx_type ppos73= patch_positions[73];idx_type ppos74= patch_positions[74];idx_type ppos75= patch_positions[75];idx_type ppos76= patch_positions[76];
			idx_type ppos77= patch_positions[77];idx_type ppos78= patch_positions[78];idx_type ppos79= patch_positions[79];idx_type ppos80= patch_positions[80];
			idx_type ppos81= patch_positions[81];idx_type ppos82= patch_positions[82];idx_type ppos83= patch_positions[83];idx_type ppos84= patch_positions[84];
			idx_type ppos85= patch_positions[85];idx_type ppos86= patch_positions[86];idx_type ppos87= patch_positions[87];idx_type ppos88= patch_positions[88];
			idx_type ppos89= patch_positions[89];idx_type ppos90= patch_positions[90];idx_type ppos91= patch_positions[91];idx_type ppos92= patch_positions[92];
			idx_type ppos93= patch_positions[93];idx_type ppos94= patch_positions[94];idx_type ppos95= patch_positions[95];idx_type ppos96= patch_positions[96];
			idx_type ppos97= patch_positions[97];idx_type ppos98= patch_positions[98];idx_type ppos99= patch_positions[99];idx_type ppos100= patch_positions[100];
			idx_type ppos101= patch_positions[101];idx_type ppos102= patch_positions[102];idx_type ppos103= patch_positions[103];idx_type ppos104= patch_positions[104];
			idx_type ppos105= patch_positions[105];idx_type ppos106= patch_positions[106];idx_type ppos107= patch_positions[107];idx_type ppos108= patch_positions[108];
			idx_type ppos109= patch_positions[109];idx_type ppos110= patch_positions[110];idx_type ppos111= patch_positions[111];idx_type ppos112= patch_positions[112];
			idx_type ppos113= patch_positions[113];idx_type ppos114= patch_positions[114];

			//get patchvalues at origin (this is the fastest approach)
			float val_orig0 = image_previous[idx0        ];float val_orig1=image_previous[idx0+ppos1];float val_orig2=image_previous[idx0+ppos2];float val_orig3=image_previous[idx0+ppos3];float val_orig4=image_previous[idx0+ppos4];
			float val_orig5=image_previous[idx0+ppos5];float val_orig6=image_previous[idx0+ppos6];float val_orig7=image_previous[idx0+ppos7];float val_orig8=image_previous[idx0+ppos8];
			float val_orig9=image_previous[idx0+ppos9];float val_orig10=image_previous[idx0+ppos10];float val_orig11=image_previous[idx0+ppos11];float val_orig12=image_previous[idx0+ppos12];
			float val_orig13=image_previous[idx0+ppos13];float val_orig14=image_previous[idx0+ppos14];float val_orig15=image_previous[idx0+ppos15];float val_orig16=image_previous[idx0+ppos16];
			float val_orig17=image_previous[idx0+ppos17];float val_orig18=image_previous[idx0+ppos18];float val_orig19=image_previous[idx0+ppos19];float val_orig20=image_previous[idx0+ppos20];
			float val_orig21=image_previous[idx0+ppos21];float val_orig22=image_previous[idx0+ppos22];float val_orig23=image_previous[idx0+ppos23];float val_orig24=image_previous[idx0+ppos24];
			float val_orig25=image_previous[idx0+ppos25];float val_orig26=image_previous[idx0+ppos26];float val_orig27=image_previous[idx0+ppos27];float val_orig28=image_previous[idx0+ppos28];
			float val_orig29=image_previous[idx0+ppos29];float val_orig30=image_previous[idx0+ppos30];float val_orig31=image_previous[idx0+ppos31];float val_orig32=image_previous[idx0+ppos32];
			float val_orig33=image_previous[idx0+ppos33];float val_orig34=image_previous[idx0+ppos34];float val_orig35=image_previous[idx0+ppos35];float val_orig36=image_previous[idx0+ppos36];
			float val_orig37=image_previous[idx0+ppos37];float val_orig38=image_previous[idx0+ppos38];float val_orig39=image_previous[idx0+ppos39];float val_orig40=image_previous[idx0+ppos40];
			float val_orig41=image_previous[idx0+ppos41];float val_orig42=image_previous[idx0+ppos42];float val_orig43=image_previous[idx0+ppos43];float val_orig44=image_previous[idx0+ppos44];
			float val_orig45=image_previous[idx0+ppos45];float val_orig46=image_previous[idx0+ppos46];float val_orig47=image_previous[idx0+ppos47];float val_orig48=image_previous[idx0+ppos48];
			float val_orig49=image_previous[idx0+ppos49];float val_orig50=image_previous[idx0+ppos50];float val_orig51=image_previous[idx0+ppos51];float val_orig52=image_previous[idx0+ppos52];
			float val_orig53=image_previous[idx0+ppos53];float val_orig54=image_previous[idx0+ppos54];float val_orig55=image_previous[idx0+ppos55];float val_orig56=image_previous[idx0+ppos56];
			float val_orig57=image_previous[idx0+ppos57];float val_orig58=image_previous[idx0+ppos58];float val_orig59=image_previous[idx0+ppos59];float val_orig60=image_previous[idx0+ppos60];
			float val_orig61=image_previous[idx0+ppos61];float val_orig62=image_previous[idx0+ppos62];float val_orig63=image_previous[idx0+ppos63];float val_orig64=image_previous[idx0+ppos64];
			float val_orig65=image_previous[idx0+ppos65];float val_orig66=image_previous[idx0+ppos66];float val_orig67=image_previous[idx0+ppos67];float val_orig68=image_previous[idx0+ppos68];
			float val_orig69=image_previous[idx0+ppos69];float val_orig70=image_previous[idx0+ppos70];float val_orig71=image_previous[idx0+ppos71];float val_orig72=image_previous[idx0+ppos72];
			float val_orig73=image_previous[idx0+ppos73];float val_orig74=image_previous[idx0+ppos74];float val_orig75=image_previous[idx0+ppos75];float val_orig76=image_previous[idx0+ppos76];
			float val_orig77=image_previous[idx0+ppos77];float val_orig78=image_previous[idx0+ppos78];float val_orig79=image_previous[idx0+ppos79];float val_orig80=image_previous[idx0+ppos80];
			float val_orig81=image_previous[idx0+ppos81];float val_orig82=image_previous[idx0+ppos82];float val_orig83=image_previous[idx0+ppos83];float val_orig84=image_previous[idx0+ppos84];
			float val_orig85=image_previous[idx0+ppos85];float val_orig86=image_previous[idx0+ppos86];float val_orig87=image_previous[idx0+ppos87];float val_orig88=image_previous[idx0+ppos88];
			float val_orig89=image_previous[idx0+ppos89];float val_orig90=image_previous[idx0+ppos90];float val_orig91=image_previous[idx0+ppos91];float val_orig92=image_previous[idx0+ppos92];
			float val_orig93=image_previous[idx0+ppos93];float val_orig94=image_previous[idx0+ppos94];float val_orig95=image_previous[idx0+ppos95];float val_orig96=image_previous[idx0+ppos96];
			float val_orig97=image_previous[idx0+ppos97];float val_orig98=image_previous[idx0+ppos98];float val_orig99=image_previous[idx0+ppos99];float val_orig100=image_previous[idx0+ppos100];
			float val_orig101=image_previous[idx0+ppos101];float val_orig102=image_previous[idx0+ppos102];float val_orig103=image_previous[idx0+ppos103];float val_orig104=image_previous[idx0+ppos104];
			float val_orig105=image_previous[idx0+ppos105];float val_orig106=image_previous[idx0+ppos106];float val_orig107=image_previous[idx0+ppos107];float val_orig108=image_previous[idx0+ppos108];
			float val_orig109=image_previous[idx0+ppos109];float val_orig110=image_previous[idx0+ppos110];float val_orig111=image_previous[idx0+ppos111];float val_orig112=image_previous[idx0+ppos112];
			float val_orig113=image_previous[idx0+ppos113];float val_orig114=image_previous[idx0+ppos114];

			float sigma = sigma_list[z0-zpad];
			float multiplier = -1.f/(sigma*sigma*beta);
			/////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////
			for (int s = 0; s < nsize_search; s++)
			{
				__syncthreads();

				idx_type idx1 = idx0 + search_positions[s];
				float noisy_value_searchpos = image_raw[idx1];

				//get patchvalues at search position
				/////////////////////////////////////////////////////////////////////////
				float distance = 0.0f;

				float tmp;
				tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos1]-val_orig1; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos2]-val_orig2; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos3]-val_orig3; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos4]-val_orig4; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos5]-val_orig5; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos6]-val_orig6; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos7]-val_orig7; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos8]-val_orig8; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos9]-val_orig9; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos10]-val_orig10; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos11]-val_orig11; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos12]-val_orig12; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos13]-val_orig13; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos14]-val_orig14; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos15]-val_orig15; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos16]-val_orig16; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos17]-val_orig17; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos18]-val_orig18; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos19]-val_orig19; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos20]-val_orig20; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos21]-val_orig21; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos22]-val_orig22; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos23]-val_orig23; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos24]-val_orig24; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos25]-val_orig25; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos26]-val_orig26; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos27]-val_orig27; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos28]-val_orig28; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos29]-val_orig29; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos30]-val_orig30; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos31]-val_orig31; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos32]-val_orig32; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos33]-val_orig33; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos34]-val_orig34; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos35]-val_orig35; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos36]-val_orig36; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos37]-val_orig37; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos38]-val_orig38; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos39]-val_orig39; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos40]-val_orig40; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos41]-val_orig41; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos42]-val_orig42; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos43]-val_orig43; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos44]-val_orig44; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos45]-val_orig45; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos46]-val_orig46; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos47]-val_orig47; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos48]-val_orig48; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos49]-val_orig49; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos50]-val_orig50; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos51]-val_orig51; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos52]-val_orig52; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos53]-val_orig53; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos54]-val_orig54; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos55]-val_orig55; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos56]-val_orig56; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos57]-val_orig57; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos58]-val_orig58; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos59]-val_orig59; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos60]-val_orig60; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos61]-val_orig61; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos62]-val_orig62; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos63]-val_orig63; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos64]-val_orig64; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos65]-val_orig65; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos66]-val_orig66; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos67]-val_orig67; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos68]-val_orig68; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos69]-val_orig69; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos70]-val_orig70; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos71]-val_orig71; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos72]-val_orig72; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos73]-val_orig73; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos74]-val_orig74; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos75]-val_orig75; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos76]-val_orig76; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos77]-val_orig77; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos78]-val_orig78; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos79]-val_orig79; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos80]-val_orig80; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos81]-val_orig81; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos82]-val_orig82; distance += (tmp*tmp)*0.111111f;
				tmp = image_previous[idx1 + ppos83]-val_orig83; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos84]-val_orig84; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos85]-val_orig85; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos86]-val_orig86; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos87]-val_orig87; distance += (tmp*tmp)*0.0682275f;
				tmp = image_previous[idx1 + ppos88]-val_orig88; distance += (tmp*tmp)*0.0501801f;
				tmp = image_previous[idx1 + ppos89]-val_orig89; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos90]-val_orig90; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos91]-val_orig91; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos92]-val_orig92; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos93]-val_orig93; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos94]-val_orig94; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos95]-val_orig95; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos96]-val_orig96; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos97]-val_orig97; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos98]-val_orig98; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos99]-val_orig99; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos100]-val_orig100; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos101]-val_orig101; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos102]-val_orig102; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos103]-val_orig103; distance += (tmp*tmp)*0.04f;
				tmp = image_previous[idx1 + ppos104]-val_orig104; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos105]-val_orig105; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos106]-val_orig106; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos107]-val_orig107; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos108]-val_orig108; distance += (tmp*tmp)*0.0333954f;
				tmp = image_previous[idx1 + ppos109]-val_orig109; distance += (tmp*tmp)*0.0287373f;
				tmp = image_previous[idx1 + ppos110]-val_orig110; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos111]-val_orig111; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos112]-val_orig112; distance += (tmp*tmp)*0.0225664f;
				tmp = image_previous[idx1 + ppos113]-val_orig113; distance += (tmp*tmp)*0.0204082f;
				tmp = image_previous[idx1 + ppos114]-val_orig114; distance += (tmp*tmp)*0.0204082f;
				/////////////////////////////////////////////////////////////////////////

				//weight the patch
				/////////////////////////////////////////////////////////////////////////
				distance = distance*multiplier;
				float this_weight = expf(distance); //this is faster on GPU than an own approximation

				filtervalue += this_weight*noisy_value_searchpos;
				filterweight += this_weight;

				maxweight = ((this_weight > maxweight) ? this_weight : maxweight);
				/////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////////////////////////////////

			if (maxweight > 0.0f)
			{
				filtervalue += maxweight*noisy_value_origin;
				filterweight += maxweight;

				filtervalue /= filterweight;
			}
			else
				filtervalue = noisy_value_origin;

			__syncthreads();
			next_result[idx_unpadded] = filtervalue;

			return;
		}
	}

	int IterativeNLM_GPU::configure_device(int shape[3], protocol::DenoiseParameters *params)
	{
		int devicecount = 0;
		hipGetDeviceCount(&devicecount);
		if(devicecount < 1) {params->gpu.n_gpus = 0; return -1;}
		else params->gpu.n_gpus = std::min(devicecount-params->gpu.deviceID, params->gpu.n_gpus);

		deviceID = params->gpu.deviceID;
		ngpus = params->gpu.n_gpus;
		threadsPerBlock = params->gpu.threadsPerBlock;
		hipSetDevice(deviceID);

		//Unrolling does not improve speed on GPU
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		padding[0] = params->radius_searchspace[0]+params->radius_patchspace[0];
		padding[1] = params->radius_searchspace[1]+params->radius_patchspace[1];
		padding[2] = std::min(params->nslices/2, params->radius_searchspace[2])+params->radius_patchspace[2];
		padding[3] = params->radius_searchspace[0]+params->radius_patchspace[0];
		padding[4] = params->radius_searchspace[1]+params->radius_patchspace[1];
		padding[5] = std::min(params->nslices/2, params->radius_searchspace[2])+params->radius_patchspace[2];

		shape_padded[0] = shape[0]+padding[0]+padding[3];
		shape_padded[1] = shape[1]+padding[1]+padding[4];
		shape_padded[2] = shape[2]+padding[2]+padding[5];
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//precalculate immutable values
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		idx_type *search_positions_host = setup_searchspace(shape_padded, params, nsize_search);
		idx_type *patch_positions_host = setup_patchspace(shape_padded, params, nsize_patch);
		float *distweight_host = setup_distweight(shape_padded, params);

		//pointer arrays for multi GPU
		search_positions = new idx_type*[params->gpu.n_gpus];
		patch_positions  = new idx_type*[params->gpu.n_gpus];
		distweight       = new float*[params->gpu.n_gpus];
		sigma_list       = new float*[params->gpu.n_gpus];

		for (int gpu = 0; gpu < params->gpu.n_gpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);

			(idx_type*) hipMalloc((void**)&search_positions[gpu], nsize_search*sizeof(*search_positions[gpu]));
			(idx_type*) hipMalloc((void**)&patch_positions[gpu], nsize_patch*sizeof(*patch_positions[gpu]));
			(float*) hipMalloc((void**)&distweight[gpu], nsize_patch*sizeof(*distweight[gpu]));
			(float*) hipMalloc((void**)&sigma_list[gpu], shape[2]*sizeof(*sigma_list[gpu]));

			hipMemcpy(search_positions[gpu], search_positions_host, nsize_search*sizeof(*search_positions[gpu]), hipMemcpyHostToDevice);
			hipMemcpy(patch_positions[gpu], patch_positions_host, nsize_patch*sizeof(*patch_positions[gpu]), hipMemcpyHostToDevice);
			hipMemcpy(distweight[gpu], distweight_host, nsize_patch*sizeof(*distweight[gpu]), hipMemcpyHostToDevice);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		//generate kernel code
		if (0 == 1){
			std::ofstream outfile;
		    outfile.open("/home/stefan/Documents/kernelcode.txt");

		    outfile << "patchsize: " << nsize_patch << "\n";

			for (int i = 0; i < nsize_patch; i++)
			{
				if (i == 0) continue;
				else outfile << "idx_type ppos"<<i<< "= patch_positions["<<i<<"];";
				if (i != 0 && (i%4) == 0) outfile << "\n";
			}
			outfile << "\n-----------------------------\n";
			for (int i = 0; i < nsize_patch; i++)
			{
				if (i == 0)outfile << "float val_orig0 = image_previous[idx0        ];";
				else outfile << "float val_orig"<<i<<"=image_previous[idx0+ppos"<<i<<"];";
				if (i != 0 && (i%4) == 0) outfile << "\n";
			}
			outfile << "\n-----------------------------\n";
			for (int i = 0; i < nsize_patch; i++)
			{
				if (i== 0)outfile << "tmp = image_previous[idx1        ]-val_orig0; distance += (tmp*tmp)*" << distweight_host[i] << "f;\n";
				else      outfile<< "tmp = image_previous[idx1 + ppos" << i << "]-val_orig"<<i<<"; distance += (tmp*tmp)*" << distweight_host[i] << "f;\n";
			}
			outfile << "\n-----------------------------\n";
			outfile.close();
		}

		free(search_positions_host);
		free(patch_positions_host);
		free(distweight_host);
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//estimate how many slices can be denoised at once
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		n_parallelslices = shape[2]/ngpus;
		if((shape[2]%ngpus) != 0) n_parallelslices++;

		long long int nslice = shape[0]*shape[1];
		long long int nstack = n_parallelslices*nslice;

		long long int nslice_padded = shape_padded[0]*shape_padded[1];
		long long int nstack_padded = (n_parallelslices+padding[2]+padding[5])*nslice_padded;

		size_t free_byte, total_byte ;
		double free_db;

		for (int gpu = 0; gpu < params->gpu.n_gpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);
			hipMemGetInfo( &free_byte, &total_byte ) ;

			if (gpu == 0)
				free_db = (double)free_byte-params->gpu.memory_buffer*free_db; //subtract some memory to be kept free
			else
				free_db = std::min(free_db, (double)free_byte-params->gpu.memory_buffer*free_db);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		double expected_usage = (2*nstack_padded+nstack)*sizeof(float);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, deviceID);

		while (expected_usage > free_db || nstack_padded*sizeof(float) > props.maxGridSize[0])
		{
			n_parallelslices--;
			nstack_padded -= nslice_padded;
			nstack -= nslice;

			expected_usage = (2*nstack_padded+nstack)*sizeof(float);
		}
		////////////////////////////////////////////////////

		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//allocate sufficient gpu-memory
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		idx_type asize1 = nstack_padded*sizeof(*image_raw[0]);
		idx_type asize2 = nstack_padded*sizeof(*image_previous[0]);
		idx_type asize3 = nstack       *sizeof(*next_result[0]);

		image_raw      = new float*[params->gpu.n_gpus];
		image_previous = new float*[params->gpu.n_gpus];
		next_result    = new float*[params->gpu.n_gpus];

		for (int gpu = 0; gpu < params->gpu.n_gpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);
			(float*) hipMalloc((void**)&image_raw[gpu], asize1*sizeof(*image_raw[gpu]));
			(float*) hipMalloc((void**)&image_previous[gpu], asize2*sizeof(*image_previous[gpu]));
			(float*) hipMalloc((void**)&next_result[gpu], asize3*sizeof(*next_result[gpu]));
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//copy immutable denoise parameters to constant memory
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int gpu = 0; gpu < params->gpu.n_gpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nslices_searchspace), &params->nslices, sizeof(gpu_const::nslices_searchspace));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::beta), &params->beta, sizeof(gpu_const::beta));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nsize_patch), &nsize_patch, sizeof(gpu_const::nsize_patch));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nsize_search), &nsize_search, sizeof(gpu_const::nsize_search));

			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::radius_searchspace), &params->radius_searchspace,  3*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::radius_patchspace), &params->radius_patchspace,  3*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::padding), &padding,  6*sizeof(int), 0);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);

			std::string error_string = (std::string) hipGetErrorString(hipGetLastError());
			if (error_string != "no error")
			{
				std::cout << "Device Configuration: " << error_string << std::endl;
				return -2;
			}
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		hipSetDevice(deviceID);
		return n_parallelslices;
	}
	void IterativeNLM_GPU::free_device()
	{
		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);

			hipFree(search_positions[gpu]);
			hipFree(patch_positions[gpu]);
			hipFree(distweight[gpu]);
			hipFree(sigma_list[gpu]);
			hipFree(image_raw[gpu]);
			hipFree(image_previous[gpu]);
			hipFree(next_result[gpu]);
		}
		hipSetDevice(deviceID);

		free(search_positions);
		free(patch_positions);
		free(distweight);
		free(sigma_list);
		free(image_raw);
		free(image_previous);
		free(next_result);

		return;
	}

	void IterativeNLM_GPU::Run_GaussianNoise(int iter, float* instack, int shape[3], protocol::DenoiseParameters *params)
	{
		// The straightforward method for small stacks:
		//     puts all slices on a single GPU and denoises in one pass

		hipSetDevice(deviceID);
		auto time0 = std::chrono::high_resolution_clock::now();

		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);

		//Check if already fully processed and just push result to device
		/////////////////////////////////////////////////////////////////////////////
		if (params->io.resume)
		{
			bool finished_block = true;
			struct stat buffer;

			for (int i = params->io.firstslice; i <= params->io.lastslice; i++)
			{
				//file exists?
				std::string filename = params->io.active_outpath + "/denoised" + aux::zfill_int2string(i,4)+".tif";
				if (stat (filename.c_str(), &buffer) != 0){finished_block = false; break;}

				//file is complete?
				std::ifstream testFile(filename.c_str(), std::ios::binary | std::ios::ate);
				const auto filesize = testFile.tellg();
				if (filesize > expected_filesize) expected_filesize = filesize;
				else if (filesize != expected_filesize){finished_block = false; break;}
			}
			if (finished_block)
			{
				hdcom::HdCommunication hdcom;
				std::vector<std::string> filelist = hdcom.GetFilelist(params->io.active_outpath, shape);
				float *result = hdcom.Get3DTifSequence_32bitPointer(filelist,shape,params->io.firstslice,params->io.lastslice);
				hipMemcpy(next_result[0], result, nstack*sizeof(*next_result[0]), hipMemcpyHostToDevice);
				hipDeviceSynchronize();
				free(result);

				if (iter == 1) prepare_iteration1(instack, shape);

				std::cout << "iteration " << iter << " read in from disk" << std::endl;
				return;
			}
			resumed = finished_block;
		}
		/////////////////////////////////////////////////////////////////////////////

		if (iter == 1) prepare_iteration1(instack, shape);
		else prepare_nextiteration(shape);

		if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 1)
			gpu_denoise::apply_filter_patch111<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 1)
			gpu_denoise::apply_filter_patch221<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 2)
			gpu_denoise::apply_filter_patch222<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 1)
			gpu_denoise::apply_filter_patch331<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 2)
			gpu_denoise::apply_filter_patch332<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 3)
			gpu_denoise::apply_filter_patch333<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 2)
			gpu_denoise::apply_filter_patch112<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 3)
			gpu_denoise::apply_filter_patch113<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0]);
		else
			gpu_denoise::apply_filter_generic<<<blocksPerGrid,threadsPerBlock>>>(image_raw[0], image_previous[0], next_result[0], sigma_list[0], search_positions[0], patch_positions[0], distweight[0]);
		hipDeviceSynchronize();

		auto time_final = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> elapsed_total = time_final-time0;
		std::cout << "iteration " << iter << " took " << elapsed_total.count() << " s                          " << std::endl;

		return;
	}
	void IterativeNLM_GPU::Run_GaussianNoise_GPUBlocks(int iter, float* instack, float * &previous, int shape[3], protocol::DenoiseParameters *params)
	{
		// The method for machines with plenty of RAM:
		//     puts all slices in RAM and splits denoising on the GPU by compute capacity
		//     (allows for multiGPU denoising)

		hipSetDevice(deviceID);
		auto time0 = std::chrono::high_resolution_clock::now();

		int blocklength = n_parallelslices;
		int n_blocks = shape[2]/blocklength;
		if ((shape[2]%blocklength) != 0) n_blocks++;

		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;

		//Check if already fully processed and just push result to device
		/////////////////////////////////////////////////////////////////////////////
		if (params->io.resume)
		{
			bool finished_block = true;
			struct stat buffer;

			for (int i = params->io.firstslice; i <= params->io.lastslice; i++)
			{
				//file exists?
				std::string filename = params->io.active_outpath + "/denoised" + aux::zfill_int2string(i,4)+".tif";
				if (stat (filename.c_str(), &buffer) != 0){finished_block = false; break;}

				//file is complete?
				std::ifstream testFile(filename.c_str(), std::ios::binary | std::ios::ate);
				const auto filesize = testFile.tellg();
				if (filesize > expected_filesize) expected_filesize = filesize;
				else if (filesize != expected_filesize){finished_block = false; break;}
			}
			if (finished_block)
			{
				hdcom::HdCommunication hdcom;
				std::vector<std::string> filelist = hdcom.GetFilelist(params->io.active_outpath, shape);
				free(previous);
				previous = hdcom.Get3DTifSequence_32bitPointer(filelist,shape,params->io.firstslice,params->io.lastslice);

				std::cout << "iteration " << iter << " read in from disk" << std::endl;
				return;
			}
			resumed = finished_block;
		}
		/////////////////////////////////////////////////////////////////////////////

		int* firstslice = new int[ngpus];
		int* lastslice = new int[ngpus];

		for (int n = 0; n < n_blocks; n+=ngpus)
		{
			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				firstslice[gpu] = std::min((n+gpu)*blocklength, shape[2]-1);
				lastslice[gpu] = std::min((n+1+gpu)*blocklength-1, shape[2]-1);
			}

			if (iter == 1) prepare_iteration1_block(instack, shape, firstslice, lastslice);
			else prepare_nextiteration_block(instack, previous, shape, firstslice, lastslice);

			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				nstack = ((lastslice[gpu]+1)-firstslice[gpu])*nslice;
				int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);

				if (blocksPerGrid > 0)
				{
					hipSetDevice(deviceID+gpu);

				if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch111<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch221<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch222<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch331<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch332<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 3)
					gpu_denoise::apply_filter_patch333<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch112<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 3)
					gpu_denoise::apply_filter_patch113<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else
					gpu_denoise::apply_filter_generic<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu], distweight[gpu]);

				}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				hipSetDevice(deviceID+gpu);

				std::string error_string = (std::string) hipGetErrorString(hipGetLastError());
				if (error_string != "no error")
				{
					std::cout << "Block " << n << ", GPU " << gpu << " section2: " << error_string << std::endl;
					return;
				}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			//get result
			///////////////////////////////////////////////////////////////////////////////////////////
			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				int blockshape[3] = {shape[0], shape[1], ((lastslice[gpu]+1)-firstslice[gpu])};

				hipSetDevice(deviceID+gpu);

				idx_type nstack_block = nslice*blockshape[2];
				idx_type asize1 = nstack_block*sizeof(*next_result[0]);

				if(asize1 > 0) hipMemcpyAsync(previous+(firstslice[gpu]*nslice),next_result[gpu], asize1, hipMemcpyDeviceToHost);
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}
			///////////////////////////////////////////////////////////////////////////////////////////

			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				hipSetDevice(deviceID+gpu);

				std::string error_string = (std::string) hipGetErrorString(hipGetLastError());
				if (error_string != "no error")
				{
					std::cout << "Block " << n << ", GPU " << gpu << ": " << error_string << std::endl;
					return;
				}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			//console output
			////////////////////////////////////////////////////////////////////////////////////////////
			auto time_final = std::chrono::high_resolution_clock::now();
			std::chrono::duration<double> elapsed_total = time_final-time0;
			std::cout << "iteration " << iter << ": " << std::min(shape[2], (n+ngpus)*n_parallelslices) << "/" << shape[2] << ", "
					<< round(elapsed_total.count()/(n+ngpus)*10.)/(10.f*n_parallelslices*ngpus)*
					  (shape[2]-std::min(shape[2], (n+ngpus)*n_parallelslices))*ngpus
					<< " s remaining          \r";
			std::cout.flush();
			////////////////////////////////////////////////////////////////////////////////////////////
		}

		free(firstslice);
		free(lastslice);

		auto time_final = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> elapsed_total = time_final-time0;
		std::cout << "iteration " << iter << " took " << elapsed_total.count() << " s                          " << std::endl;

		return;
	}
	void IterativeNLM_GPU::Run_GaussianNoise_GPUBlocks(int iter, std::vector<std::string> &filelist_raw, std::vector<std::string> &filelist_prev, int shape[3], protocol::DenoiseParameters *params)
	{
		//The method for insufficient memory:
		//    reads in what is necessary to fully occupy the GPU
		//

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx), &shape[0], sizeof(gpu_const::nx));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny), &shape[1], sizeof(gpu_const::ny));
		}

		//check if only substack processing and get offset in filelists
		/////////////////////////////////////////////////////////////////
		int firstslice_offset[2] = {0, 0};

		if (params->io.firstslice != 0)
		{
			firstslice_offset[0] = params->io.firstslice;

			std::string fs_sub = aux::zfill_int2string(params->io.firstslice,4)+".tif";

			for (int i = 0; i < filelist_prev.size(); i++)
				if (filelist_prev[i].substr(filelist_prev[i].length()-8,8) == fs_sub){firstslice_offset[1] = i; break;}
		}
		/////////////////////////////////////////////////////////////////

		//set the block dimensions
		/////////////////////////////////////////////////////////////////
		int blocklength = n_parallelslices;
		int n_blocks = shape[2]/blocklength;
		if ((shape[2]%blocklength) != 0) n_blocks++;

		padding[0] = params->radius_searchspace[0]+params->radius_patchspace[0];
		padding[1] = params->radius_searchspace[1]+params->radius_patchspace[1];
		padding[2] = std::min(params->nslices/2, params->radius_searchspace[2])+params->radius_patchspace[2];
		padding[3] = params->radius_searchspace[0]+params->radius_patchspace[0];
		padding[4] = params->radius_searchspace[1]+params->radius_patchspace[1];
		padding[5] = std::min(params->nslices/2, params->radius_searchspace[2])+params->radius_patchspace[2];

		int zpadding[2] = {padding[2], padding[5]};

		int blockshape[3] = {shape[0], shape[1], blocklength*ngpus};
		long long int blockslice = blockshape[0]*blockshape[1];
		long long int blockstack = blockshape[2]*blockslice;

		int blockshape_padded[3] = {blockshape[0]+padding[0]+padding[3], blockshape[1]+padding[1]+padding[4], blockshape[2]+padding[2]+padding[5]};
		idx_type blockslice_padded = blockshape_padded[0]*blockshape_padded[1];
		/////////////////////////////////////////////////////////////////

		float* output = (float*) malloc(blockstack*sizeof(*output));

		hdcom::HdCommunication hdcom;
		auto time0 = std::chrono::high_resolution_clock::now();
		int skipped_blocks = 0;

		for (int n = 0; n < n_blocks; n+=ngpus)
		{
			int firstslice = n*blocklength-zpadding[0];
			int lastslice = std::min((n+ngpus)*blocklength-1, shape[2]-1)+zpadding[1];

			//Check if block already fully processed and resume
			/////////////////////////////////////////////////////////////////////////////
			if (params->io.resume)
			{
				bool finished_block = true;
				struct stat buffer;

				for (int i = firstslice+zpadding[0]; i <= lastslice-zpadding[1]; i++)
				{
					//file exists?
					std::string filename = params->io.active_outpath + "/denoised" + aux::zfill_int2string(i+params->io.firstslice,4)+".tif";
					if (stat (filename.c_str(), &buffer) != 0){finished_block = false; break;}

					//file is complete?
					std::ifstream testFile(filename.c_str(), std::ios::binary | std::ios::ate);
					const auto filesize = testFile.tellg();
					if (filesize > expected_filesize) expected_filesize = filesize;
					else if (filesize != expected_filesize){finished_block = false; break;}
				}
				if (finished_block){skipped_blocks++; continue;}
			}
			/////////////////////////////////////////////////////////////////////////////

			//check if we need to pad in z-direction
			/////////////////////////////////////////////////////////////////////////////
			if (firstslice < 0) {padding[2] = -firstslice; firstslice = 0;}
			else padding[2] = 0;
			if (lastslice >= shape[2]) {padding[5] = lastslice-(shape[2]-1); lastslice = shape[2]-1;}
			else padding[5] = 0;
			/////////////////////////////////////////////////////////////////////////////

			//apply padding on host
			/////////////////////////////////////////////////////////////////////////////
			float *imageblock_raw, *imageblock_previous;

			if (iter == 1 && lastslice > firstslice)
				imageblock_raw = hdcom.Get3DTifSequence_32bitPointer(filelist_raw, blockshape, firstslice+firstslice_offset[0], lastslice+firstslice_offset[0]);
			else if (lastslice > firstslice)
			{
				float* tmp = hdcom.Get3DTifSequence_32bitPointer(filelist_prev, blockshape, firstslice+firstslice_offset[1], lastslice+firstslice_offset[1]);
				imageblock_previous = pad_reflective(tmp, padding, blockshape, blockshape_padded);

				free(tmp);

				imageblock_raw = hdcom.Get3DTifSequence_32bitPointer(filelist_raw, blockshape, firstslice+firstslice_offset[0], lastslice+firstslice_offset[0]);
			}
			if (lastslice > firstslice)
			{
				float* tmp = pad_reflective(imageblock_raw, padding, blockshape, blockshape_padded);

				std::swap(tmp, imageblock_raw);
				free(tmp);
			}
			/////////////////////////////////////////////////////////////////////////////

			//prepare device
			/////////////////////////////////////////////////////////////////////////////
			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				int blockdim2 = std::min((n+gpu+1)*blocklength, shape[2])-((n+gpu)*blocklength);
				long long int asize1 = ((blockdim2+zpadding[0]+zpadding[1])*blockslice_padded)*sizeof(*image_raw[gpu]);

				hipSetDevice(deviceID+gpu);
				hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &blockdim2, sizeof(gpu_const::nz));

				if(iter == 1){
					if(asize1 > 0)
					hipMemcpyAsync(image_raw[gpu], imageblock_raw+(gpu*blocklength)*blockslice_padded, asize1, hipMemcpyHostToDevice);}
				else{
					if(asize1 > 0){
					hipMemcpyAsync(image_raw[gpu], imageblock_raw+(gpu*blocklength)*blockslice_padded, asize1, hipMemcpyHostToDevice);
					hipMemcpyAsync(image_previous[gpu], imageblock_previous+(gpu*blocklength)*blockslice_padded, asize1, hipMemcpyHostToDevice);}}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			if (iter == 1){
				for (int gpu = 0; gpu < ngpus; gpu++)
				{
					int blockdim2 = std::min((n+gpu+1)*blocklength, shape[2])-((n+gpu)*blocklength);
					long long int asize1 = ((blockdim2+zpadding[0]+zpadding[1])*blockslice_padded)*sizeof(*image_raw[gpu]);

					hipSetDevice(deviceID+gpu);
					if(asize1 > 0) hipMemcpyAsync(image_previous[gpu], image_raw[gpu], asize1, hipMemcpyDeviceToDevice);
				}
				for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}
			}

			if (lastslice > firstslice && iter > 1)free(imageblock_previous);
			if (lastslice > firstslice)free(imageblock_raw);
			/////////////////////////////////////////////////////////////////////////////

			//filter on device
			/////////////////////////////////////////////////////////////////////////////
			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				int blockdim2 = std::min((n+gpu+1)*blocklength, shape[2])-((n+gpu)*blocklength);
				idx_type blockstack = (blockdim2*blockslice);
				int blocksPerGrid = (blockstack + threadsPerBlock - 1) / (threadsPerBlock);

				if (blocksPerGrid > 0)
				{
					hipSetDevice(deviceID+gpu);

				if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch111<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch221<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 2 && params->radius_patchspace[1] == 2 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch222<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 1)
					gpu_denoise::apply_filter_patch331<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch332<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 3 && params->radius_patchspace[1] == 3 && params->radius_patchspace[2] == 3)
					gpu_denoise::apply_filter_patch333<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 2)
					gpu_denoise::apply_filter_patch112<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				if (params->radius_patchspace[0] == 1 && params->radius_patchspace[1] == 1 && params->radius_patchspace[2] == 3)
					gpu_denoise::apply_filter_patch113<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu]);
				else
					gpu_denoise::apply_filter_generic<<<blocksPerGrid,threadsPerBlock>>>(image_raw[gpu], image_previous[gpu], next_result[gpu], sigma_list[gpu], search_positions[gpu], patch_positions[gpu], distweight[gpu]);
				}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			for (int gpu = 0; gpu < ngpus; gpu++)
			{
				int blockdim2 = std::min((n+gpu+1)*blocklength, shape[2])-((n+gpu)*blocklength);

				if(gpu*blocklength < shape[2] && blockdim2 > 0)
				{
					hipSetDevice(deviceID+gpu);

					idx_type this_nstack = blockslice*blockdim2;
					idx_type asize1 = this_nstack*sizeof(*next_result[gpu]);

					hipMemcpyAsync(output+(gpu*blocklength)*blockslice,next_result[gpu], asize1, hipMemcpyDeviceToHost);
				}
			}
			for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

			/////////////////////////////////////////////////////////////////////////////
			blockshape[2] = std::min(blocklength*ngpus, params->io.lastslice+1-((n*blocklength)+params->io.firstslice));

			if(params->io.save_type == "16bit") hdcom.SaveTifSequence_as16bit((n*blocklength)+params->io.firstslice, output, blockshape, params->io.active_outpath, "denoised", false);
			else hdcom.SaveTifSequence_32bit((n*blocklength)+params->io.firstslice, output, blockshape, params->io.active_outpath, "denoised", false);

			//console output
			////////////////////////////////////////////////////////////////////////////////////////////
			auto time_final = std::chrono::high_resolution_clock::now();
			std::chrono::duration<double> elapsed_total = time_final-time0;
			std::cout << "iteration " << iter << ": " << std::min(shape[2], (n+ngpus)*n_parallelslices) << "/" << shape[2] << ", "
					<< round(elapsed_total.count()/(n+ngpus-skipped_blocks)*10.)/(10.f*n_parallelslices*ngpus)*
					  (shape[2]-std::min(shape[2]-skipped_blocks*blocklength, (n+ngpus)*n_parallelslices-skipped_blocks*blocklength))*ngpus
					<< " s remaining          \r";
			std::cout.flush();
			////////////////////////////////////////////////////////////////////////////////////////////
		}
		//////////////////////////////////////////////////////////////////////////////
		free(output);

		auto time_final = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> elapsed_total = time_final-time0;
		std::cout << "iteration " << iter << " took " << elapsed_total.count() << " s                          " << std::endl;

		return;
	}

	void IterativeNLM_GPU::prepare_iteration1(float* input, int shape[3])
	{
		//
		//overwrites next_result and applies padding!
		//

		//////////////////////////////////////////////////////////////////////
		long long int nslice = shape[0]*shape[1];
		long long int nstack = shape[2]*nslice;
		long long int asize1 = nstack*sizeof(*next_result[0]);

		hipSetDevice(deviceID);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx), &shape[0], sizeof(gpu_const::nx));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny), &shape[1], sizeof(gpu_const::ny));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &shape[2], sizeof(gpu_const::nz));

		hipMemcpy(next_result[0], input, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		//////////////////////////////////////////////////////////////////////

		//////////////////////////////////////////////////////////////////////
		long long int nslice_padded = (shape[0]+padding[0]+padding[3])*(shape[1]+padding[1]+padding[4]);
		long long int nstack_padded = (shape[2]+padding[2]+padding[5])*nslice_padded;

		int blocksPerGrid = (nstack_padded + threadsPerBlock - 1) / (threadsPerBlock);

		gpu_denoise::pad_reflective<<<blocksPerGrid,threadsPerBlock>>>(next_result[0], image_raw[0]);
		hipDeviceSynchronize();
		//////////////////////////////////////////////////////////////////////

		asize1 = nstack_padded*sizeof(*image_previous[0]);
		hipMemcpy(image_previous[0], image_raw[0], asize1, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();

		return;
	}
	void IterativeNLM_GPU::prepare_iteration1_block(float* input, int shape[3], const int* firstslice, const int* lastslice)
	{
		idx_type nslice = shape[0]*shape[1];
		long long int nslice_padded = (shape[0]+padding[0]+padding[3])*(shape[1]+padding[1]+padding[4]);

		int* block_dim2 = new int[ngpus];
		int* zpadding = new int[2*ngpus];
		int* initialdepth = new int[ngpus];

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			initialdepth[gpu] = (lastslice[gpu]+1)-firstslice[gpu];
			int this_padding[6] = {padding[0], padding[1], padding[2], padding[3], padding[4], padding[5]};

			//check how we need to pad in z
			//////////////////////////////////////////////////////////////////////
			int active_firstslice = firstslice[gpu]-padding[2];
			int active_lastslice = lastslice[gpu] + padding[5];

			if (active_firstslice < 0) {this_padding[2] = -active_firstslice; active_firstslice = 0;}
			else this_padding[2] = 0;

			if (active_lastslice >= shape[2]) {this_padding[5] = active_lastslice-(shape[2]-1); active_lastslice = shape[2]-1;}
			else this_padding[5] = 0;

			zpadding[2*gpu] = this_padding[2];
			zpadding[2*gpu+1] = this_padding[5];

			block_dim2[gpu] = (active_lastslice+1)-active_firstslice;

			idx_type offset = active_firstslice*nslice;
			idx_type nstack = block_dim2[gpu]*nslice;
			long long int asize1 = nstack*sizeof(*next_result[gpu]);
			//////////////////////////////////////////////////////////////////////

			hipSetDevice(deviceID+gpu);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &block_dim2[gpu], sizeof(gpu_const::nz));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx), &shape[0], sizeof(gpu_const::nx));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny), &shape[1], sizeof(gpu_const::ny));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::padding), &this_padding,  6*sizeof(int), 0); //update padding
			if (asize1 > 0)
				hipMemcpyAsync(next_result[gpu], input+offset, asize1, hipMemcpyHostToDevice);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			long long int nstack_padded = (block_dim2[gpu]+zpadding[2*gpu]+zpadding[2*gpu+1])*nslice_padded;
			int blocksPerGrid = (nstack_padded + threadsPerBlock - 1) / (threadsPerBlock);

			hipSetDevice(deviceID+gpu);
			if (blocksPerGrid > 0)
				gpu_denoise::pad_reflective<<<blocksPerGrid,threadsPerBlock>>>(next_result[gpu], image_raw[gpu]);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			long long int nstack_padded = (block_dim2[gpu]+zpadding[2*gpu]+zpadding[2*gpu+1])*nslice_padded;
			long long int asize1 = nstack_padded*sizeof(*image_previous[gpu]);

			hipSetDevice(deviceID+gpu);
			if(asize1 > 0) hipMemcpyAsync(image_previous[gpu], image_raw[gpu], asize1, hipMemcpyDeviceToDevice);

			//reset
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &initialdepth[gpu], sizeof(gpu_const::nz));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::padding), &padding,  6*sizeof(int), 0);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		free(block_dim2);
		free(zpadding);
		free(initialdepth);

		hipSetDevice(deviceID);
		return;
	}
	void IterativeNLM_GPU::prepare_nextiteration(int shape[3])
	{
		//////////////////////////////////////////////////////////////////////
		long long int nslice_padded = (shape[0]+padding[0]+padding[3])*(shape[1]+padding[1]+padding[4]);
		long long int nstack_padded = (shape[2]+padding[2]+padding[5])*nslice_padded;

		int blocksPerGrid = (nstack_padded + threadsPerBlock - 1) / (threadsPerBlock);

		hipSetDevice(deviceID);
		gpu_denoise::pad_reflective<<<blocksPerGrid,threadsPerBlock>>>(next_result[0], image_previous[0]);
		hipDeviceSynchronize();
		//////////////////////////////////////////////////////////////////////
		return;
	}
	void IterativeNLM_GPU::prepare_nextiteration_block(float* input, float *prev_result, int shape[3], const int* firstslice, const int* lastslice)
	{
		idx_type nslice = shape[0]*shape[1];
		long long int nslice_padded = (shape[0]+padding[0]+padding[3])*(shape[1]+padding[1]+padding[4]);

		int* block_dim2 = new int[ngpus];
		int* zpadding = new int[2*ngpus];
		int* initialdepth = new int[ngpus];
		long long int* offset = new long long int[ngpus];

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			initialdepth[gpu] = (lastslice[gpu]+1)-firstslice[gpu];
			int this_padding[6] = {padding[0], padding[1], padding[2], padding[3], padding[4], padding[5]};

			//check how we need to pad in z
			//////////////////////////////////////////////////////////////////////
			int active_firstslice = firstslice[gpu]-padding[2];
			int active_lastslice = lastslice[gpu] + padding[5];

			if (active_firstslice < 0) {this_padding[2] = -active_firstslice; active_firstslice = 0;}
			else this_padding[2] = 0;

			if (active_lastslice >= shape[2]) {this_padding[5] = active_lastslice-(shape[2]-1); active_lastslice = shape[2]-1;}
			else this_padding[5] = 0;

			zpadding[2*gpu] = this_padding[2];
			zpadding[2*gpu+1] = this_padding[5];

			block_dim2[gpu] = (active_lastslice+1)-active_firstslice;

			offset[gpu] = active_firstslice*nslice;
			idx_type nstack = block_dim2[gpu]*nslice;
			long long int asize1 = nstack*sizeof(*next_result[gpu]);
			//////////////////////////////////////////////////////////////////////

			hipSetDevice(deviceID+gpu);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &block_dim2[gpu], sizeof(gpu_const::nz));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx), &shape[0], sizeof(gpu_const::nx));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny), &shape[1], sizeof(gpu_const::ny));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::padding), &this_padding,  6*sizeof(int), 0); //update padding

			if (asize1 > 0)
				hipMemcpyAsync(next_result[gpu], input+offset[gpu], asize1, hipMemcpyHostToDevice);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			long long int nstack_padded = (block_dim2[gpu]+zpadding[2*gpu]+zpadding[2*gpu+1])*nslice_padded;
			int blocksPerGrid = (nstack_padded + threadsPerBlock - 1) / (threadsPerBlock);

			hipSetDevice(deviceID+gpu);
			if (blocksPerGrid > 0)
				gpu_denoise::pad_reflective<<<blocksPerGrid,threadsPerBlock>>>(next_result[gpu], image_raw[gpu]);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			idx_type nstack = block_dim2[gpu]*nslice;
			long long int asize1 = nstack*sizeof(*next_result[gpu]);

			hipSetDevice(deviceID+gpu);
			hipMemcpyAsync(next_result[gpu], prev_result+offset[gpu], asize1, hipMemcpyHostToDevice);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			long long int nstack_padded = (block_dim2[gpu]+zpadding[2*gpu]+zpadding[2*gpu+1])*nslice_padded;
			int blocksPerGrid = (nstack_padded + threadsPerBlock - 1) / (threadsPerBlock);

			hipSetDevice(deviceID+gpu);
			if (blocksPerGrid > 0)
				gpu_denoise::pad_reflective<<<blocksPerGrid,threadsPerBlock>>>(next_result[gpu], image_previous[gpu]);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			//reset
			hipSetDevice(deviceID+gpu);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz), &initialdepth[gpu], sizeof(gpu_const::nz));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::padding), &padding,  6*sizeof(int), 0);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		free(block_dim2);
		free(zpadding);
		free(initialdepth);
		free(offset);
		hipSetDevice(deviceID);

		return;
	}

	void IterativeNLM_GPU::set_sigma(float* sigmalist, int shape[3])
	{
		int nz = shape[2];
		long long int asize1 = nz*sizeof(*sigma_list[0]);

		for (int gpu = 0; gpu < ngpus; gpu++)
		{
			hipSetDevice(deviceID+gpu);
			hipMemcpyAsync(sigma_list[gpu], sigmalist, asize1, hipMemcpyHostToDevice);
		}
		for (int gpu = 0; gpu < ngpus; gpu++){hipSetDevice(deviceID+gpu);hipDeviceSynchronize();}

		hipSetDevice(deviceID);
		return;
	}
	void IterativeNLM_GPU::get_result(float* output, int shape[3])
	{
		hipSetDevice(deviceID);

		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = nstack*sizeof(*next_result[0]);

		hipMemcpy(output,next_result[0], asize1, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		return;
	}
}
